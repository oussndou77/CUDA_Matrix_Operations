#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Fonctions CPU
void MatrixInit(float *M, int n, int p);
void MatrixPrint(float *M, int n, int p);
void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p);
void MatrixMult(float *M1, float *M2, float *Mout, int n);

// Fonctions GPU
__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p);
__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n);

int main() {
    int n = 1000; // Taille des matrices carrées
    size_t size = n * n * sizeof(float);

    // Allocation mémoire sur CPU
    float *h_M1 = (float *)malloc(size);
    float *h_M2 = (float *)malloc(size);
    float *h_Mout = (float *)malloc(size);

    // Initialisation des matrices
    MatrixInit(h_M1, n, n);
    MatrixInit(h_M2, n, n);

    printf("Matrice 1 :\n");
    MatrixPrint(h_M1, n, n);

    printf("Matrice 2 :\n");
    MatrixPrint(h_M2, n, n);

    // Addition sur CPU
    MatrixAdd(h_M1, h_M2, h_Mout, n, n);
    printf("Resultat de l'addition (CPU) :\n");
    MatrixPrint(h_Mout, n, n);

    // Multiplication sur CPU
    MatrixMult(h_M1, h_M2, h_Mout, n);
    printf("Resultat de la multiplication (CPU) :\n");
    MatrixPrint(h_Mout, n, n);

    // Allocation mémoire sur GPU
    float *d_M1, *d_M2, *d_Mout;
    hipMalloc((void **)&d_M1, size);
    hipMalloc((void **)&d_M2, size);
    hipMalloc((void **)&d_Mout, size);

    // Transfert des données CPU -> GPU
    hipMemcpy(d_M1, h_M1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M2, h_M2, size, hipMemcpyHostToDevice);

    // Configuration des dimensions du grid et du block
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Addition sur GPU
    cudaMatrixAdd<<<numBlocks, threadsPerBlock>>>(d_M1, d_M2, d_Mout, n, n);
    hipMemcpy(h_Mout, d_Mout, size, hipMemcpyDeviceToHost);
    printf("Resultat de l'addition (GPU) :\n");
    MatrixPrint(h_Mout, n, n);

    // Multiplication sur GPU
    cudaMatrixMult<<<numBlocks, threadsPerBlock>>>(d_M1, d_M2, d_Mout, n);
    // Synchronisation et transfert des résultats GPU -> CPU
    hipMemcpy(h_Mout, d_Mout, size, hipMemcpyDeviceToHost);
    // Affichage des résultats
    printf("Resultat de la multiplication (GPU) :\n");
    MatrixPrint(h_Mout, n, n);

    // Libération de la mémoire
    free(h_M1);
    free(h_M2);
    free(h_Mout);
    hipFree(d_M1);
    hipFree(d_M2);
    hipFree(d_Mout);

    return 0;
}

// Fonction pour initialiser une matrice avec des valeurs aléatoires entre -1 et 1
void MatrixInit(float *M, int n, int p) {
    for (int i = 0; i < n * p; i++) {
        M[i] = (float)(rand() % 200 - 100) / 100.0;
    }
}

// Fonction pour afficher une matrice
void MatrixPrint(float *M, int n, int p) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            printf("%.2f ", M[i * p + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Fonction pour additionner deux matrices sur CPU
void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    for (int i = 0; i < n * p; i++) {
        Mout[i] = M1[i] + M2[i];
    }
}

// Fonction pour multiplier deux matrices sur CPU
void MatrixMult(float *M1, float *M2, float *Mout, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            Mout[i * n + j] = 0;
            for (int k = 0; k < n; k++) {
                Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
            }
        }
    }
}

// Kernel pour additionner deux matrices sur GPU
__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < p) {
        int index = row * p + col;
        Mout[index] = M1[index] + M2[index];
    }
}

// Kernel pour multiplier deux matrices sur GPU
__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0;
        for (int k = 0; k < n; k++) {
            sum += M1[row * n + k] * M2[k * n + col];
        }
        Mout[row * n + col] = sum;
    }
}
